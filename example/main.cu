#include <iostream>
#include <hip/hip_runtime.h>


__global__ void helloCUDA() {
    printf("Hello from CUDA Kernel!\n\n");
}


int main() {
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
